
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void vectorAdd(float *A,float *B,float *C,int n){
int i = blockDim.x*blockIdx.x+threadIdx.x;
if (i<n) C[i] = A[i]+B[i];
}
int main(){
const int n =10;
float A[n],B[n],C[n];
float *d_a,*d_b,*d_c;
hipMalloc(&d_a,n*sizeof(float));
hipMalloc(&d_b,n*sizeof(float));
hipMalloc(&d_c,n*sizeof(float));
hipMemcpy(d_a,A,n*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(d_b,B,n*sizeof(float),hipMemcpyHostToDevice);
int blockSize = 256;
int gridSize = ceil(n/blockSize);
vectorAdd<<<gridSize,blockSize>>>(d_a,d_b,d_c,n);
hipMemcpy(d_c,C,n*sizeof(float),hipMemcpyDeviceToHost);
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

}